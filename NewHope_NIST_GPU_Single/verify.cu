
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdint.h>

/*************************************************
* Name:        verify
* 
* Description: Compare two arrays for equality in constant time.
*
* Arguments:    unsigned char *a: pointer to first byte array
*               unsigned char *b: pointer to second byte array
*              size_t len:             length of the byte arrays
*
* Returns 0 if the byte arrays are equal, 1 otherwise
**************************************************/
int verify( unsigned char* a,  unsigned char* b, size_t len)
{
	//uint64_t r;
	size_t i;
	//r = 0;

	for (i = 0; i < len; i++)
	{
		if (a[i] != b[i])
			return 1;
	}

	return 0;
}

/*************************************************
* Name:        cmov
* 
* Description: Copy len bytes from x to r if b is 1;
*              don't modify x if b is 0. Requires b to be in {0,1};
*              assumes two's complement representation of negative integers.
*              Runs in constant time.
*
* Arguments:   unsigned char *r:       pointer to output byte array
*               unsigned char *x: pointer to input byte array
*              size_t len:             Amount of bytes to be copied
*              unsigned char b:        Condition bit; has to be in {0,1}
**************************************************/
void cmov(unsigned char *r,  unsigned char *x, size_t len, unsigned char b)
{
  size_t i;

  b = -b;
  for(i=0;i<len;i++)
    r[i] ^= b & (x[i] ^ r[i]);
}
