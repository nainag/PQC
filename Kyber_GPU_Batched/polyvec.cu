#include "hip/hip_runtime.h"

// @Author: Arpan Jati
// Adapted from NewHope Reference Codebase and Parallelized using CUDA
// Updated : August 2019

#include <stdint.h>
#include "polyvec.h"
#include "poly.h"

/*************************************************
* Name:        polyvec_compress
*
* Description: Compress and serialize vector of polynomials
*
* Arguments:   - unsigned char *r: pointer to output byte array (needs space for KYBER_POLYVECCOMPRESSEDBYTES)
*              -  polyvec *a: pointer to input vector of polynomials
**************************************************/
__device__ void polyvec_compress(unsigned char* r, polyvec* a)
{
	int i, j, k;

	int X = threadIdx.x + blockIdx.x * blockDim.x;

	polyvec_csubq(a);

	uint16_t t[8];
	for (i = 0; i < KYBER_K; i++)
	{
		for (j = 0; j < KYBER_N / 8; j++)
		{
			for (k = 0; k < 8; k++)
			{
				t[k] = ((((uint32_t)a->vec[i].coeffs[8 * j + k].threads[X] << 11) + KYBER_Q / 2) / KYBER_Q) & 0x7ff;
			}

			r[11 * j + 0] = t[0] & 0xff;
			r[11 * j + 1] = (t[0] >> 8) | ((t[1] & 0x1f) << 3);
			r[11 * j + 2] = (t[1] >> 5) | ((t[2] & 0x03) << 6);
			r[11 * j + 3] = (t[2] >> 2) & 0xff;
			r[11 * j + 4] = (t[2] >> 10) | ((t[3] & 0x7f) << 1);
			r[11 * j + 5] = (t[3] >> 7) | ((t[4] & 0x0f) << 4);
			r[11 * j + 6] = (t[4] >> 4) | ((t[5] & 0x01) << 7);
			r[11 * j + 7] = (t[5] >> 1) & 0xff;
			r[11 * j + 8] = (t[5] >> 9) | ((t[6] & 0x3f) << 2);
			r[11 * j + 9] = (t[6] >> 6) | ((t[7] & 0x07) << 5);
			r[11 * j + 10] = (t[7] >> 3);
		}
		r += 352;
	}
}

/*************************************************
* Name:        polyvec_decompress
*
* Description: De-serialize and decompress vector of polynomials;
*              approximate inverse of polyvec_compress
*
* Arguments:   - polyvec *r:       pointer to output vector of polynomials
*              - unsigned char *a: pointer to input byte array (of length KYBER_POLYVECCOMPRESSEDBYTES)
**************************************************/
__device__ void polyvec_decompress(polyvec* r,  unsigned char* a)
{
	int i, j;

	int X = threadIdx.x + blockIdx.x * blockDim.x;

	for (i = 0; i < KYBER_K; i++)
	{
		for (j = 0; j < KYBER_N / 8; j++)
		{
			r->vec[i].coeffs[8 * j + 0].threads[X] = (((a[11 * j + 0] | (((uint32_t)a[11 * j + 1] & 0x07) << 8)) * KYBER_Q) + 1024) >> 11;
			r->vec[i].coeffs[8 * j + 1].threads[X] = ((((a[11 * j + 1] >> 3) | (((uint32_t)a[11 * j + 2] & 0x3f) << 5)) * KYBER_Q) + 1024) >> 11;
			r->vec[i].coeffs[8 * j + 2].threads[X] = ((((a[11 * j + 2] >> 6) | (((uint32_t)a[11 * j + 3] & 0xff) << 2) | (((uint32_t)a[11 * j + 4] & 0x01) << 10)) * KYBER_Q) + 1024) >> 11;
			r->vec[i].coeffs[8 * j + 3].threads[X] = ((((a[11 * j + 4] >> 1) | (((uint32_t)a[11 * j + 5] & 0x0f) << 7)) * KYBER_Q) + 1024) >> 11;
			r->vec[i].coeffs[8 * j + 4].threads[X] = ((((a[11 * j + 5] >> 4) | (((uint32_t)a[11 * j + 6] & 0x7f) << 4)) * KYBER_Q) + 1024) >> 11;
			r->vec[i].coeffs[8 * j + 5].threads[X] = ((((a[11 * j + 6] >> 7) | (((uint32_t)a[11 * j + 7] & 0xff) << 1) | (((uint32_t)a[11 * j + 8] & 0x03) << 9)) * KYBER_Q) + 1024) >> 11;
			r->vec[i].coeffs[8 * j + 6].threads[X] = ((((a[11 * j + 8] >> 2) | (((uint32_t)a[11 * j + 9] & 0x1f) << 6)) * KYBER_Q) + 1024) >> 11;
			r->vec[i].coeffs[8 * j + 7].threads[X] = ((((a[11 * j + 9] >> 5) | (((uint32_t)a[11 * j + 10] & 0xff) << 3)) * KYBER_Q) + 1024) >> 11;
		}
		a += 352;
	}
}

/*************************************************
* Name:        polyvec_tobytes
*
* Description: Serialize vector of polynomials
*
* Arguments:   - unsigned char *r: pointer to output byte array (needs space for KYBER_POLYVECBYTES)
*              -  polyvec *a: pointer to input vector of polynomials
**************************************************/
__device__ void polyvec_tobytes(unsigned char* r, polyvec* a)
{
	int i;
	for (i = 0; i < KYBER_K; i++)
	{
		poly_tobytes(r + i * KYBER_POLYBYTES, &a->vec[i]);
	}
}

/*************************************************
* Name:        polyvec_frombytes
*
* Description: De-serialize vector of polynomials;
*              inverse of polyvec_tobytes
*
* Arguments:   - unsigned char *r: pointer to output byte array
*              -  polyvec *a: pointer to input vector of polynomials (of length KYBER_POLYVECBYTES)
**************************************************/
__device__ void polyvec_frombytes(polyvec* r,  unsigned char* a)
{
	int i;
	for (i = 0; i < KYBER_K; i++)
		poly_frombytes(&r->vec[i], a + i * KYBER_POLYBYTES);
}

/*************************************************
* Name:        polyvec_ntt
*
* Description: Apply forward NTT to all elements of a vector of polynomials
*
* Arguments:   - polyvec *r: pointer to in/output vector of polynomials
**************************************************/
__global__ void polyvec_ntt_n(int COUNT, polyvec* r)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		int i;
		for (i = 0; i < KYBER_K; i++)
			poly_ntt(&r->vec[i]);
	}
}

/*************************************************
* Name:        polyvec_invntt
*
* Description: Apply inverse NTT to all elements of a vector of polynomials
*
* Arguments:   - polyvec *r: pointer to in/output vector of polynomials
**************************************************/
__global__ void polyvec_invntt_n(int COUNT, polyvec* r)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		for (int i = 0; i < KYBER_K; i++)
			poly_invntt(&r->vec[i]);
	}
}

/*************************************************
* Name:        polyvec_pointwise_acc
*
* Description: Pointwise multiply elements of a and b and accumulate into r
*
* Arguments: - poly *r:          pointer to output polynomial
*            -  polyvec *a: pointer to first input vector of polynomials
*            -  polyvec *b: pointer to second input vector of polynomials
**************************************************/
__global__ void polyvec_pointwise_acc_n(int COUNT, poly* r, polyvec* a, polyvec* b, poly* temp)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		int i;
		poly* t = temp;

		poly_basemul(r, &a->vec[0], &b->vec[0]);
		for (i = 1; i < KYBER_K; i++) {
			poly_basemul(t, &a->vec[i], &b->vec[i]);
			poly_add(r, r, t);
		}

		poly_reduce(r);
	}
}

/*************************************************
* Name:        polyvec_reduce
*
* Description: Applies Barrett reduction to each coefficient
*              of each element of a vector of polynomials
*              for details of the Barrett reduction see comments in reduce.c
*
* Arguments:   - poly *r:       pointer to input/output polynomial
**************************************************/
__device__  void polyvec_reduce(polyvec* r)
{
	//int X = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = 0; i < KYBER_K; i++)
		poly_reduce(&r->vec[i]);

}

__global__  void polyvec_reduce_n(int COUNT, polyvec* r)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		for (int i = 0; i < KYBER_K; i++)
			poly_reduce(&r->vec[i]);
	}
}

/*************************************************
* Name:        polyvec_csubq
*
* Description: Applies conditional subtraction of q to each coefficient
*              of each element of a vector of polynomials
*              for details of conditional subtraction of q see comments in reduce.c
*
* Arguments:   - poly *r:       pointer to input/output polynomial
**************************************************/
__device__ void polyvec_csubq(polyvec* r)
{
	for (int i = 0; i < KYBER_K; i++)
	{
		poly_csubq(&r->vec[i]);
	}
}

/*************************************************
* Name:        polyvec_add
*
* Description: Add vectors of polynomials
*
* Arguments: - polyvec *r:       pointer to output vector of polynomials
*            -  polyvec *a: pointer to first input vector of polynomials
*            -  polyvec *b: pointer to second input vector of polynomials
**************************************************/
__global__  void polyvec_add_n(int COUNT, polyvec* r,  polyvec* a,  polyvec* b)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		for (int i = 0; i < KYBER_K; i++)
			poly_add(&r->vec[i], &a->vec[i], &b->vec[i]);
	}
}

__device__  void polyvec_add(int COUNT, polyvec* r,  polyvec* a,  polyvec* b)
{
	for (int i = 0; i < KYBER_K; i++)
		poly_add(&r->vec[i], &a->vec[i], &b->vec[i]);
}
