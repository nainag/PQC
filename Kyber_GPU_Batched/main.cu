
// @Author: Arpan Jati
// Adapted from NewHope Reference Codebase and Parallelized using CUDA
// Updated : August 2019
// -------------------------------------------------------------
// CODE FOR PERFORMANCE COMPARISON. NOT FOR ACTUAL DEPLOYMENT
// -------------------------------------------------------------

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include "rng.h"
#include "api.h"

#define	MAX_MARKER_LEN		50
#define KAT_SUCCESS          0
#define KAT_FILE_OPEN_ERROR -1
#define KAT_DATA_ERROR      -3
#define KAT_CRYPTO_FAILURE  -4


/*

int main_ind_cca()
{
	int                 ret_val;

	unsigned char ct[CRYPTO_CIPHERTEXTBYTES];
	unsigned char ss[CRYPTO_BYTES];
	unsigned char ss1[CRYPTO_BYTES];

	unsigned char pk[CRYPTO_PUBLICKEYBYTES];
	unsigned char sk[CRYPTO_SECRETKEYBYTES];

	ret_val = crypto_kem_keypair(pk, sk);

	if (ret_val != 0) {
		printf("crypto_kem_keypair returned <%d>\n", ret_val);
		return KAT_CRYPTO_FAILURE;
	}

	ret_val = crypto_kem_enc(ct, ss, pk);

	if (ret_val != 0) {
		printf("crypto_kem_enc returned <%d>\n", ret_val);
		return KAT_CRYPTO_FAILURE;
	}

	ret_val = crypto_kem_dec(ss1, ct, sk);

	if (ret_val != 0) {
		printf("crypto_kem_dec returned <%d>\n", ret_val);
		return KAT_CRYPTO_FAILURE;
	}

	if (memcmp(ss, ss1, CRYPTO_BYTES)) {
		printf("crypto_kem_dec returned bad 'ss' value\n");
		return KAT_CRYPTO_FAILURE;
	}

	return 0;
}
*/

#include "params.h"
#include "indcpa.h"
#include "main.h"

#include <chrono>
#include <iostream>

using namespace std;

// MAIN CPA 

void allocatePolySet(poly_set4* polySet)
{
	HANDLE_ERROR(hipMalloc(&(polySet->a), sizeof(poly)));
	HANDLE_ERROR(hipMalloc(&(polySet->b), sizeof(poly)));
	HANDLE_ERROR(hipMalloc(&(polySet->c), sizeof(poly)));
	HANDLE_ERROR(hipMalloc(&(polySet->d), sizeof(poly)));

	HANDLE_ERROR(hipMalloc(&(polySet->av), sizeof(polyvec)));
	HANDLE_ERROR(hipMalloc(&(polySet->bv), sizeof(polyvec)));
	HANDLE_ERROR(hipMalloc(&(polySet->cv), sizeof(polyvec)));
	HANDLE_ERROR(hipMalloc(&(polySet->dv), sizeof(polyvec)));
	HANDLE_ERROR(hipMalloc(&(polySet->ev), sizeof(polyvec)));
	HANDLE_ERROR(hipMalloc(&(polySet->fv), sizeof(polyvec)));

	HANDLE_ERROR(hipMalloc(&(polySet->AV), sizeof(polyvec) * 4));

	HANDLE_ERROR(hipMalloc(&(polySet->seed), (KYBER_SYMBYTES * 2) * N_TESTS));

	HANDLE_ERROR(hipMalloc(&(polySet->large_buffer_a), LARGE_BUFFER_SZ * N_TESTS));
	HANDLE_ERROR(hipMalloc(&(polySet->large_buffer_b), LARGE_BUFFER_SZ * N_TESTS));
}

void freePolySet(poly_set4* polySet)
{
	HANDLE_ERROR(hipFree(polySet->a));
	HANDLE_ERROR(hipFree(polySet->b));
	HANDLE_ERROR(hipFree(polySet->c));
	HANDLE_ERROR(hipFree(polySet->d));

	HANDLE_ERROR(hipFree(polySet->av));
	HANDLE_ERROR(hipFree(polySet->bv));
	HANDLE_ERROR(hipFree(polySet->cv));
	HANDLE_ERROR(hipFree(polySet->dv));
	HANDLE_ERROR(hipFree(polySet->ev));
	HANDLE_ERROR(hipFree(polySet->fv));

	HANDLE_ERROR(hipFree(polySet->seed));

	HANDLE_ERROR(hipFree(polySet->large_buffer_a));
	HANDLE_ERROR(hipFree(polySet->large_buffer_b));
}


int Hardware()
{
	hipDeviceProp_t prop;
	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		printf(" --- General Information for device %d ---\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device copy overlap: ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execition timeout : ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf(" --- Memory Information for device %d ---\n", i);
		printf("Total global mem: %llu\n", prop.totalGlobalMem);
		printf("Total constant Mem: %llu\n", prop.totalConstMem);
		printf("Max mem pitch: %zd\n", prop.memPitch);
		printf("Texture Alignment: %zd\n", prop.textureAlignment);
		printf(" --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count: %d\n", prop.multiProcessorCount);

		MP_COUNT = prop.multiProcessorCount;

		printf("Shared mem per mp: %zd\n", prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");
	}

	return count;
}

int SELECTED_GPU = GPU_G1060;

#ifdef ANALYSIS_MODE
int NORMAL_COUNTS[4] = { 1024,4096,16384,32768 };
#else
int NORMAL_COUNTS[15] = { 4,8,16,32,64,128,256,512,1024,2048,4096,8192,16384,20000,32768 };
#endif

int COUNT = N_TESTS;

int main()
{
	printf("\n KYBER GPU BATCHED | N_TESTS = %d ", N_TESTS);

	printf("\nN_TESTS: %d \n", N_TESTS);

	int gpu_count = Hardware();

	printf("\n SELECT GPU TYPE (for optimization): ");
	printf("\n		GPU_G1060 = 0");
	printf("\n		GPU_P6000 = 1");
	printf("\n		GPU_940MX = 2");
	printf("\n		GPU_V100  = 3");
	printf("\n		    (0-3) = ");

	char buffer[20];

	auto str = fgets(buffer, 20, stdin);
	//auto str = "3";

	if (str != NULL)
	{
		int v = atoi(str);

		if (v < 0 || v >3)
		{
			printf("\n NO_SUCH_GPU !!");
			exit(1);
		}
		else
		{
			SELECTED_GPU = v;
		}
	}
	else
	{
		printf("\n INVALID INPUT !!");
		exit(1);
	}

	printf("\n SELECTED GPU %s \n\n", SELECTED_GPU_NAME);

	hipEvent_t start, stop;

	int SELECTED_GPU_ID = 0;

	if (gpu_count > 1)
	{
		printf("\nSELECT GPU ID (for execution) (0-%d): ", (gpu_count - 1));
		auto str = fgets(buffer, 20, stdin);

		if (str != NULL)
		{
			int v = atoi(str);

			if (v < 0 || v > 3)
			{
				printf("\n NO_SUCH_GPU_ID !!");
				exit(1);
			}
			else
			{
				SELECTED_GPU_ID = v;
			}
		}
	}

	printf("\n SELECTED GPU ID = %d \n", SELECTED_GPU_ID);

	// Choose which GPU to run on, change this on a multi-GPU system.
	HANDLE_ERROR(hipSetDevice(SELECTED_GPU_ID));

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//////////////////////////////

	printf("\n SERVER MODE (TYPE '1' to enable): ");

	int serverMode = 0;

	auto ch = getchar();

	//auto ch = '0';
	if (ch == '1')
	{
		serverMode = 1;
		printf("\n SERVER MODE ENABLED [A-C]");
		printf("\n // -------------------------------------------------------------------");
		printf("\n // NOTE THAT FOR SERVER MODE, THE RESULTS WILL BE INCORRECT !! ");
		printf("\n // FURTHER, THIS IS NOT THE EXACT PERFORMANCE AS WE EMULATE THE SERVER BEHAVIOR");
		printf("\n // AND COMPUTE TOTAL AMOUNT OF COMPUTATIONS FOR A SATURATED SERVER.");
		printf("\n // Keeping 'SERVERMODE = 0' performs all THREE stages of the key exchange together.");
		printf("\n // Not realistic ! But, lets us know the performance improvements by comparing GPU");
		printf("\n // results with a CPU based implementation.");
		printf("\n // -------------------------------------------------------------------");
	}
	else
	{
		printf("\n SERVER MODE DISABLED [A-B-C]");
	}

	// int N_TESTS_D = (N_TESTS);

	poly_set4 tempPoly_0[4];
	poly_set4 tempPoly_1[4];

	allocatePolySet(&tempPoly_0[0]);
	allocatePolySet(&tempPoly_1[0]);

	unsigned char* pk_h_0;
	unsigned char* sk_h_0;
	unsigned char* ct_h_0;
	unsigned char* msg1_h_0;
	unsigned char* msg2_h_0;
	unsigned char* coins_h_0;
	unsigned char* rng_buf_h_0;

	unsigned char* pk_h_1;
	unsigned char* sk_h_1;
	unsigned char* ct_h_1;
	unsigned char* msg1_h_1;
	unsigned char* msg2_h_1;
	unsigned char* coins_h_1;
	unsigned char* rng_buf_h_1;

	HANDLE_ERROR(hipHostAlloc((void**)&pk_h_0, KYBER_INDCPA_PUBLICKEYBYTES * N_TESTS, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&sk_h_0, KYBER_INDCPA_SECRETKEYBYTES * N_TESTS, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&ct_h_0, KYBER_INDCPA_BYTES * N_TESTS, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&msg1_h_0, KYBER_INDCPA_MSGBYTES * N_TESTS, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&msg2_h_0, KYBER_INDCPA_MSGBYTES * N_TESTS, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&coins_h_0, KYBER_SYMBYTES * N_TESTS, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&rng_buf_h_0, KYBER_SYMBYTES * 2 * N_TESTS, hipHostMallocDefault));

	HANDLE_ERROR(hipHostAlloc((void**)&pk_h_1, KYBER_INDCPA_PUBLICKEYBYTES * N_TESTS, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&sk_h_1, KYBER_INDCPA_SECRETKEYBYTES * N_TESTS, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&ct_h_1, KYBER_INDCPA_BYTES * N_TESTS, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&msg1_h_1, KYBER_INDCPA_MSGBYTES * N_TESTS, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&msg2_h_1, KYBER_INDCPA_MSGBYTES * N_TESTS, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&coins_h_1, KYBER_SYMBYTES * N_TESTS, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&rng_buf_h_1, KYBER_SYMBYTES * 2 * N_TESTS, hipHostMallocDefault));

	//
	unsigned char* pk_d_0;
	unsigned char* sk_d_0;
	unsigned char* ct_d_0;
	unsigned char* msg1_d_0;
	unsigned char* msg2_d_0;
	unsigned char* coins_d_0;
	unsigned char* rng_buf_d_0;


	unsigned char* pk_d_1;
	unsigned char* sk_d_1;
	unsigned char* ct_d_1;
	unsigned char* msg1_d_1;
	unsigned char* msg2_d_1;
	unsigned char* coins_d_1;
	unsigned char* rng_buf_d_1;

	HANDLE_ERROR(hipMalloc((void**)&pk_d_0, KYBER_INDCPA_PUBLICKEYBYTES * N_TESTS));
	HANDLE_ERROR(hipMalloc((void**)&sk_d_0, KYBER_INDCPA_SECRETKEYBYTES * N_TESTS));
	HANDLE_ERROR(hipMalloc((void**)&ct_d_0, KYBER_INDCPA_BYTES * N_TESTS));
	HANDLE_ERROR(hipMalloc((void**)&msg1_d_0, KYBER_INDCPA_MSGBYTES * N_TESTS));
	HANDLE_ERROR(hipMalloc((void**)&msg2_d_0, KYBER_INDCPA_MSGBYTES * N_TESTS));
	HANDLE_ERROR(hipMalloc((void**)&coins_d_0, KYBER_SYMBYTES * N_TESTS));
	HANDLE_ERROR(hipMalloc((void**)&rng_buf_d_0, KYBER_SYMBYTES * 2 * N_TESTS));

	HANDLE_ERROR(hipMalloc((void**)&pk_d_1, KYBER_INDCPA_PUBLICKEYBYTES * N_TESTS));
	HANDLE_ERROR(hipMalloc((void**)&sk_d_1, KYBER_INDCPA_SECRETKEYBYTES * N_TESTS));
	HANDLE_ERROR(hipMalloc((void**)&ct_d_1, KYBER_INDCPA_BYTES * N_TESTS));
	HANDLE_ERROR(hipMalloc((void**)&msg1_d_1, KYBER_INDCPA_MSGBYTES * N_TESTS));
	HANDLE_ERROR(hipMalloc((void**)&msg2_d_1, KYBER_INDCPA_MSGBYTES * N_TESTS));
	HANDLE_ERROR(hipMalloc((void**)&coins_d_1, KYBER_SYMBYTES * N_TESTS));
	HANDLE_ERROR(hipMalloc((void**)&rng_buf_d_1, KYBER_SYMBYTES * 2 * N_TESTS));

	memset(msg1_h_0, 0, KYBER_SYMBYTES * N_TESTS);
	randombytes(msg1_h_0, KYBER_SYMBYTES * N_TESTS);
	randombytes(coins_h_0, KYBER_SYMBYTES * N_TESTS);
	randombytes(rng_buf_h_0, KYBER_SYMBYTES * N_TESTS * 2);

	memset(msg1_h_1, 0, KYBER_SYMBYTES * N_TESTS);
	randombytes(msg1_h_1, KYBER_SYMBYTES * N_TESTS);
	randombytes(coins_h_1, KYBER_SYMBYTES * N_TESTS);
	randombytes(rng_buf_h_1, KYBER_SYMBYTES * N_TESTS * 2);


	hipStream_t stream_0;
	hipStream_t stream_1;

	HANDLE_ERROR(hipStreamCreate(&stream_0));
	HANDLE_ERROR(hipStreamCreate(&stream_1));

	// stream_1 = stream_0;

#ifdef ANALYSIS_MODE
	for (int i = 0; i < 4; i++)
#else
	for (int i = 0; i < 13; i++)
#endif
	{
		COUNT = NORMAL_COUNTS[i];

		hipEventRecord(start);

		HANDLE_ERROR(hipMemcpyAsync(pk_d_0, pk_h_0, KYBER_INDCPA_PUBLICKEYBYTES * COUNT, hipMemcpyHostToDevice, stream_0));
		HANDLE_ERROR(hipMemcpyAsync(sk_d_0, sk_h_0, KYBER_INDCPA_SECRETKEYBYTES * COUNT, hipMemcpyHostToDevice, stream_0));
		HANDLE_ERROR(hipMemcpyAsync(ct_d_0, ct_h_0, KYBER_INDCPA_BYTES * COUNT, hipMemcpyHostToDevice, stream_0));
		HANDLE_ERROR(hipMemcpyAsync(msg1_d_0, msg1_h_0, KYBER_INDCPA_MSGBYTES * COUNT, hipMemcpyHostToDevice, stream_0));
		HANDLE_ERROR(hipMemcpyAsync(msg2_d_0, msg2_h_0, KYBER_INDCPA_MSGBYTES * COUNT, hipMemcpyHostToDevice, stream_0));
		HANDLE_ERROR(hipMemcpyAsync(coins_d_0, coins_h_0, KYBER_SYMBYTES * COUNT, hipMemcpyHostToDevice, stream_0));
		HANDLE_ERROR(hipMemcpyAsync(rng_buf_d_0, rng_buf_h_0, KYBER_SYMBYTES * 2 * COUNT, hipMemcpyHostToDevice, stream_0));

		HANDLE_ERROR(hipMemcpyAsync(pk_d_1, pk_h_1, KYBER_INDCPA_PUBLICKEYBYTES * COUNT, hipMemcpyHostToDevice, stream_1));
		HANDLE_ERROR(hipMemcpyAsync(sk_d_1, sk_h_1, KYBER_INDCPA_SECRETKEYBYTES * COUNT, hipMemcpyHostToDevice, stream_1));
		HANDLE_ERROR(hipMemcpyAsync(ct_d_1, ct_h_1, KYBER_INDCPA_BYTES * COUNT, hipMemcpyHostToDevice, stream_1));
		HANDLE_ERROR(hipMemcpyAsync(msg1_d_1, msg1_h_1, KYBER_INDCPA_MSGBYTES * COUNT, hipMemcpyHostToDevice, stream_1));
		HANDLE_ERROR(hipMemcpyAsync(msg2_d_1, msg2_h_1, KYBER_INDCPA_MSGBYTES * COUNT, hipMemcpyHostToDevice, stream_1));
		HANDLE_ERROR(hipMemcpyAsync(coins_d_1, coins_h_1, KYBER_SYMBYTES * COUNT, hipMemcpyHostToDevice, stream_1));
		HANDLE_ERROR(hipMemcpyAsync(rng_buf_d_1, rng_buf_h_1, KYBER_SYMBYTES * 2 * COUNT, hipMemcpyHostToDevice, stream_1));

		indcpa_keypair(COUNT, &tempPoly_0[0], pk_d_0, sk_d_0, rng_buf_d_0, stream_0);
		indcpa_keypair(COUNT, &tempPoly_1[0], pk_d_1, sk_d_1, rng_buf_d_1, stream_1);

		if (!serverMode)
		{
			indcpa_enc(COUNT, &tempPoly_0[0], ct_d_0, msg1_d_0, pk_d_0, coins_d_0, stream_0);
			indcpa_enc(COUNT, &tempPoly_1[0], ct_d_1, msg1_d_1, pk_d_1, coins_d_1, stream_1);
		}

		indcpa_dec(COUNT, &tempPoly_0[0], msg2_d_0, ct_d_0, sk_d_0, stream_0);
		indcpa_dec(COUNT, &tempPoly_1[0], msg2_d_1, ct_d_1, sk_d_1, stream_1);

		HANDLE_ERROR(hipMemcpyAsync(pk_h_0, pk_d_0, KYBER_INDCPA_PUBLICKEYBYTES * COUNT, hipMemcpyDeviceToHost, stream_0));
		HANDLE_ERROR(hipMemcpyAsync(sk_h_0, sk_d_0, KYBER_INDCPA_SECRETKEYBYTES * COUNT, hipMemcpyDeviceToHost, stream_0));
		HANDLE_ERROR(hipMemcpyAsync(ct_h_0, ct_d_0, KYBER_INDCPA_BYTES * COUNT, hipMemcpyDeviceToHost, stream_0));
		HANDLE_ERROR(hipMemcpyAsync(msg1_h_0, msg1_d_0, KYBER_INDCPA_MSGBYTES * COUNT, hipMemcpyDeviceToHost, stream_0));
		HANDLE_ERROR(hipMemcpyAsync(msg2_h_0, msg2_d_0, KYBER_INDCPA_MSGBYTES * COUNT, hipMemcpyDeviceToHost, stream_0));
		HANDLE_ERROR(hipMemcpyAsync(coins_h_0, coins_d_0, KYBER_SYMBYTES * COUNT, hipMemcpyDeviceToHost, stream_0));
		HANDLE_ERROR(hipMemcpyAsync(rng_buf_h_0, rng_buf_d_0, KYBER_SYMBYTES * 2 * COUNT, hipMemcpyDeviceToHost, stream_0));

		HANDLE_ERROR(hipMemcpyAsync(pk_h_1, pk_d_1, KYBER_INDCPA_PUBLICKEYBYTES * COUNT, hipMemcpyDeviceToHost, stream_1));
		HANDLE_ERROR(hipMemcpyAsync(sk_h_1, sk_d_1, KYBER_INDCPA_SECRETKEYBYTES * COUNT, hipMemcpyDeviceToHost, stream_1));
		HANDLE_ERROR(hipMemcpyAsync(ct_h_1, ct_d_1, KYBER_INDCPA_BYTES * COUNT, hipMemcpyDeviceToHost, stream_1));
		HANDLE_ERROR(hipMemcpyAsync(msg1_h_1, msg1_d_1, KYBER_INDCPA_MSGBYTES * COUNT, hipMemcpyDeviceToHost, stream_1));
		HANDLE_ERROR(hipMemcpyAsync(msg2_h_1, msg2_d_1, KYBER_INDCPA_MSGBYTES * COUNT, hipMemcpyDeviceToHost, stream_1));
		HANDLE_ERROR(hipMemcpyAsync(coins_h_1, coins_d_1, KYBER_SYMBYTES * COUNT, hipMemcpyDeviceToHost, stream_1));
		HANDLE_ERROR(hipMemcpyAsync(rng_buf_h_1, rng_buf_d_1, KYBER_SYMBYTES * 2 * COUNT, hipMemcpyDeviceToHost, stream_1));

		hipEventRecord(stop);
		hipEventSynchronize(stop);

		///////////////////

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);

		auto KXps = (int)((((double)(COUNT * 2)) * (double)1000.0) / (double)milliseconds);

		printf("\n =================--------------================------------==================");
		printf("\n COUNT=%d | Time Elapsed: %f ms. K/s: %d ", COUNT * 2, milliseconds, KXps);
		printf("\n =================--------------================------------================== \n");


		int error = -1;
		int match_count = 0;

		for (int i = 0; i < KYBER_INDCPA_MSGBYTES * COUNT; i++)
		{
			if (msg1_h_0[i] != msg2_h_0[i])
			{
				error = i;
				break;
			}
		}

		for (int i = 0; i < KYBER_INDCPA_MSGBYTES * COUNT; i++)
		{
			if (msg1_h_0[i] == msg2_h_0[i])
			{
				match_count++;
			}
		}

		int max_match = KYBER_INDCPA_MSGBYTES * COUNT;

		if (error != -1)
		{
			printf(" ERROR : < MESSAGE VERIFICATION #0 > \n IDX: %d \n INST-ID: %d \n ERROR-COUNT: %d \n MAX-MATCH: %d \n MATCH-RATIO: %f\n",
				error, (error / KYBER_INDCPA_MSGBYTES), max_match - match_count, max_match, (match_count * 100.0F) / max_match);
		}
		else
		{
			printf(" MESSAGE VERIFICATION COMPLETE #0 !! \n");
		}

		error = -1;
		match_count = 0;

		for (int i = 0; i < KYBER_INDCPA_MSGBYTES * COUNT; i++)
		{
			if (msg1_h_1[i] != msg2_h_1[i])
			{
				error = i;
				break;
			}
		}

		for (int i = 0; i < KYBER_INDCPA_MSGBYTES * COUNT; i++)
		{
			if (msg1_h_1[i] == msg2_h_1[i])
			{
				match_count++;
			}
		}

		max_match = KYBER_INDCPA_MSGBYTES * COUNT;

		if (error != -1)
		{
			printf(" ERROR : < MESSAGE VERIFICATION #1 > \n IDX: %d \n INST-ID: %d \n ERROR-COUNT: %d \n MAX-MATCH: %d \n MATCH-RATIO: %f\n",
				error, (error / KYBER_INDCPA_MSGBYTES), max_match - match_count, max_match, (match_count * 100.0F) / max_match);
		}
		else
		{
			printf(" MESSAGE VERIFICATION COMPLETE #1 !! \n");
		}

		printf(" =================--------------================------------================== \n");

	}

	/*
		print_data("PK", pk_h, KYBER_INDCPA_PUBLICKEYBYTES);
		print_data("SK", sk_h, KYBER_INDCPA_SECRETKEYBYTES);
		print_data("CT", ct_h, KYBER_INDCPA_BYTES);

		print_data("msg1", msg1_h, KYBER_INDCPA_MSGBYTES);
		print_data("msg2", msg2_h + KYBER_INDCPA_MSGBYTES, KYBER_INDCPA_MSGBYTES);

		print_data("coins", coins_h, KYBER_SYMBYTES);
	*/

	// printf("\n\n");

	//print_data("msg1_0", msg1_h_0, KYBER_INDCPA_MSGBYTES);
	//print_data("msg2_0", msg2_h_0, KYBER_INDCPA_MSGBYTES);

	//print_data("msg1_1", msg1_h_1, KYBER_INDCPA_MSGBYTES);
	//print_data("msg2_1", msg2_h_1, KYBER_INDCPA_MSGBYTES);


	/////////////////////////////////////////

	HANDLE_ERROR(hipStreamDestroy(stream_0));
	HANDLE_ERROR(hipStreamDestroy(stream_1));

	HANDLE_ERROR(hipFree(pk_d_0));
	HANDLE_ERROR(hipFree(sk_d_0));
	HANDLE_ERROR(hipFree(ct_d_0));
	HANDLE_ERROR(hipFree(msg1_d_0));
	HANDLE_ERROR(hipFree(msg2_d_0));
	HANDLE_ERROR(hipFree(coins_d_0));
	HANDLE_ERROR(hipFree(rng_buf_d_0));

	HANDLE_ERROR(hipFree(pk_d_1));
	HANDLE_ERROR(hipFree(sk_d_1));
	HANDLE_ERROR(hipFree(ct_d_1));
	HANDLE_ERROR(hipFree(msg1_d_1));
	HANDLE_ERROR(hipFree(msg2_d_1));
	HANDLE_ERROR(hipFree(coins_d_1));
	HANDLE_ERROR(hipFree(rng_buf_d_1));

	HANDLE_ERROR(hipHostFree(pk_h_0));
	HANDLE_ERROR(hipHostFree(sk_h_0));
	HANDLE_ERROR(hipHostFree(ct_h_0));
	HANDLE_ERROR(hipHostFree(msg1_h_0));
	HANDLE_ERROR(hipHostFree(msg2_h_0));
	HANDLE_ERROR(hipHostFree(coins_h_0));
	HANDLE_ERROR(hipHostFree(rng_buf_h_0));

	HANDLE_ERROR(hipHostFree(pk_h_1));
	HANDLE_ERROR(hipHostFree(sk_h_1));
	HANDLE_ERROR(hipHostFree(ct_h_1));
	HANDLE_ERROR(hipHostFree(msg1_h_1));
	HANDLE_ERROR(hipHostFree(msg2_h_1));
	HANDLE_ERROR(hipHostFree(coins_h_1));
	HANDLE_ERROR(hipHostFree(rng_buf_h_1));

	freePolySet(&tempPoly_0[0]);
	freePolySet(&tempPoly_1[0]);


	// Check for any errors launching the kernel
	HANDLE_ERROR(hipGetLastError());

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	HANDLE_ERROR(hipDeviceSynchronize());


	HANDLE_ERROR(hipDeviceReset());

	printf("\n\nDONE. ");

	//getchar();

	return 0;
}